#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <math.h>
#include <windows.h>
#include "EasyBMP_1.06/EasyBMP.h"


#define BLOCK_SIZE 128
#define W 1000
#define H W



struct Pixel {
    int r, g, b;
};

struct Material {
    double diffuse, specular_c, specular_k;
};

struct Sphere {
    double pos[3];
    double r;
    double color[3];
    Material mat;
};

struct Light {
    double pos[3];
    double color[3];
    double ambient;
};

struct Camera {
    double pos[3];
    double pt[3];
};

__device__ double dot(double* a, double* b) {
    return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}

__device__ double intersectSphere(double* O, double* ray_dir, Sphere* s) {
    double a = dot(ray_dir, ray_dir);
    double OS[] = {O[0]- s->pos[0], O[1]- s->pos[1], O[2]- s->pos[2] };
    double b = 2.0 * dot(ray_dir, OS);
    double c = dot(OS, OS) - s->r * s->r;
    double disc = b * b - 4 * a * c;

    if (disc > 0.0) {
        double discSqrt = sqrt(disc);
        double q = b < 0.0 ? (-b - discSqrt) / 2.0 : (-b + discSqrt) / 2.0;
        double t0 = q / a;
        double t1 = c / q;
        if (t0 > t1) {
            double temp = t0;
            t0 = t1;
            t1 = temp;
        }
        if (t1 >= 0) {
            if (t0 < 0) {
                return t1;
            }
            else {
                return t0;
            }
        }
    }
    return DBL_MAX / 2.0;
}

__device__ double* normalizeVec3(double* a) {
    double* vec = new double[3];
    double len = sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);

    vec[0] = a[0] / len;
    vec[1] = a[1] / len;
    vec[2] = a[2] / len;

    return vec;
}

__device__ double clampColor(double x) {
    if (x > 1.0) return 1.0;
    if (x < 0.0) return 0.0;
    return x;
}

__global__ void kernel(Pixel* colors, Sphere* sphere, Light* light, Camera* camera) {
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= W * H) return;

    int h = idx / W;
    int w = idx % W;
 
    double pos_x = (2.0 * w) / W - 1.0;
    double pos_y = (2.0 * h) / H - 1.0;

    double _ray_dir[3] = { camera->pt[0] - pos_x, camera->pt[1] - pos_y, camera->pt[2] - camera->pos[2] };
    double* ray_dir = normalizeVec3(_ray_dir);


    double t = intersectSphere(camera->pos, ray_dir, sphere);

    if (t > DBL_MAX / 2.0 - 1.0) {
        colors[idx].r = 0.0;
        colors[idx].g = 0.0;
        colors[idx].b = 0.0;
        return;
    }


    double M[] = {camera->pos[0] + ray_dir[0] * t, camera->pos[1] + ray_dir[1] * t, camera->pos[2] + ray_dir[2] * t, };
    double _N[] = {M[0] - sphere->pos[0], M[1] - sphere->pos[1], M[2] - sphere->pos[2]};
    double* N = normalizeVec3(_N);
    double _toL[] = {light->pos[0] - M[0], light->pos[1] - M[1], light->pos[2] - M[2]};
    double* toL = normalizeVec3(_toL);
    double _toO[] = {camera->pos[0] -M[0], camera->pos[1]-M[1], camera->pos[2] -M[2] };
    double* toO = normalizeVec3(_toO);

    double col[] = {light->ambient, light->ambient,light->ambient};
    double _col = sphere->mat.diffuse * (dot(N, toL) > 0.0 ? dot(N, toL) : 0.0);
    
    col[0] += _col * sphere->color[0];
    col[1] += _col * sphere->color[1];
    col[2] += _col * sphere->color[2];


    double _normSum[] = {toL[0] + toO[0], toL[1] + toO[1], toL[2] + toO[2] };
    double* normSum = normalizeVec3(_normSum);

    double _spM = dot(N, normSum);
    double spM = pow(_spM > 0.0 ? _spM : 0.0, sphere->mat.specular_k) * sphere->mat.specular_c;

    col[0] += spM * light->color[0];
    col[1] += spM * light->color[1];
    col[2] += spM * light->color[2];

    colors[idx].r = clampColor(col[0]) * 255;
    colors[idx].g = clampColor(col[1]) * 255;
    colors[idx].b = clampColor(col[2]) * 255;


    delete ray_dir;
    delete N;
    delete toL;
    delete toO;
    delete normSum;
}


int main() {
    Pixel* d_colors;
    hipMalloc(&d_colors, W * H * sizeof(Pixel));
    Pixel* colors = (Pixel*)malloc(W * H * sizeof(Pixel));

    BMP Output;
    Output.SetSize(W, H);
    Output.SetBitDepth(24);

    Sphere* d_sphere1;
    Sphere sphere1 = {
        0.0, 1.0, 1.5,
        1.0,
        0.0, 0.0, 1.0,
        1.0, 1.0, 50.0
    };

    Light* d_light1;
    Light light1 = {
        5.0, 3.0, -10.0,
        1.0, 1.0, 1.0,
        0.05
    };

    Camera* d_camera;
    Camera camera = {
        0.0, 1.0, -2.0,
        0.0, 0.0, 0.0
    };

    hipMalloc(&d_sphere1, sizeof(Sphere));
    hipMemcpy(d_sphere1, &sphere1, sizeof(Sphere), hipMemcpyHostToDevice);

    hipMalloc(&d_light1, sizeof(Light));
    hipMemcpy(d_light1, &light1, sizeof(Light), hipMemcpyHostToDevice);

    hipMalloc(&d_camera, sizeof(Camera));
    hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);

    LARGE_INTEGER frequency;
    LARGE_INTEGER start;
    LARGE_INTEGER end;
    double interval;
    QueryPerformanceFrequency(&frequency);
    QueryPerformanceCounter(&start);

    kernel << <(W * H + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >> > (d_colors, d_sphere1, d_light1, d_camera);
    hipMemcpy(colors, d_colors, W * H * sizeof(Pixel), hipMemcpyDeviceToHost);

    QueryPerformanceCounter(&end);
    interval = (double)(end.QuadPart - start.QuadPart) / frequency.QuadPart;

    printf("[GPU] Time to render the image: %fs\n", interval);
    



    for (int i = 0; i < W * H; ++i) {
        RGBApixel temp;
        temp.Red = colors[i].r;
        temp.Blue = colors[i].b;
        temp.Green = colors[i].g;
        Output.SetPixel(i % W, i / W, temp);
    }


    free(colors);
    hipFree(d_colors);
    hipFree(d_sphere1);
    hipFree(d_light1);
    hipFree(d_camera);
    
    Output.WriteToFile("gpu.bmp");

    return 0;
}